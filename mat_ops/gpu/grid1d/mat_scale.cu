#include "hip/hip_runtime.h"
#include <chrono>
#include "array2d.h"
#include "cuda_helper.h"
#include "mat_bench.h"

#define value_t float
#define index_t int

__global__ void kernel(index_t Nx, index_t Ny, value_t a, value_t *x)
{
    // grid moves along last index first
    // int N_grid_j = (Ny + blockDim.y - 1) / blockDim.y;
    // int grid_i = blockIdx.x / N_grid_j;
    // int grid_j = blockIdx.x - grid_i * N_grid_j;

    // grid moves along first index first
    int N_grid_i = (Nx + blockDim.x - 1) / blockDim.x;
    int grid_j = blockIdx.x / N_grid_i;
    int grid_i = blockIdx.x - grid_j * N_grid_i;

    int i = grid_i * blockDim.x + threadIdx.x;
    int j = grid_j * blockDim.y + threadIdx.y;
    int gid = i * Ny + j;
    if (i < Nx && j < Ny)
        x[gid] *= a;
}

struct mat_copy : public mat_bench<value_t, index_t>
{
    void benchmark()
    {
        print_bench();

        std::cout << "\nSimulation info: 2d mat scale\n";

        value_t **x = create_array2d<value_t, index_t>(side_size, side_size);

#pragma omp parallel for
        for (index_t i = 0; i < side_size; i++)
        {
            for (index_t j = 0; j < side_size; j++)
            {
                x[i][j] = 1.0;
            }
        }

        value_t *d_x;
        value_t *h_x = x[0];
        checkCudaErrors(hipMalloc(&d_x, total_size * sizeof(value_t)));

        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        checkCudaErrors(hipMemcpy(d_x, h_x, total_size * sizeof(value_t), hipMemcpyHostToDevice));

        dim3 blockd3 = dim3(block0, block1, 1);
        dim3 grid = calc_grid2d2(blockd3, side_size, side_size);
        std::cout << "  Block: " << blockd3.x << "(x) X " << blockd3.y << "(y)\n"
                  << "  Grid size: " << grid.x << "\n\n";

        loops = 0;
        auto startcpu = std::chrono::high_resolution_clock::now();
        checkCudaErrors(hipEventRecord(start));
        while ((std::chrono::duration_cast<std::chrono::milliseconds>(
                    std::chrono::high_resolution_clock::now() - startcpu)
                    .count()) < 1000.0 * benchtime)
        {
            kernel<<<grid, blockd3>>>(side_size, side_size, 0.5, d_x);
            kernel<<<grid, blockd3>>>(side_size, side_size, 2.0, d_x);
            checkCudaErrorsAfterKernels;
            loops++;
        }
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float du = 0;
        checkCudaErrors(hipEventElapsedTime(&du, start, stop));
        duration = 1.0e-3 * du;

        checkCudaErrors(hipMemcpy(h_x, d_x, total_size * sizeof(value_t), hipMemcpyDeviceToHost));

        test_result(x, value_t(total_size));
        print_performance();

        delete[] x;
        checkCudaErrors(hipFree(d_x));
    }

    mat_copy(int narg, char **arg) : mat_bench<value_t, index_t>(narg, arg)
    {
        memory_transfer_per_loop = 2.0 * sizeof(value_t) * 2.0 * double(total_size) /
                                   (1024.0 * 1024.0 * 1024.0);
    }
};

int main(int narg, char **arg)
{
    check_cuda_device();
    mat_copy test(narg, arg);
    test.benchmark();
}