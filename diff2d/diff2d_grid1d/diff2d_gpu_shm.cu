#include "hip/hip_runtime.h"
#include <chrono>
#include "diff2d.h"
#include "cuda_helper.h"

#define value_t float
#define index_t int

// constants
__constant__ value_t c_zero, c_one, c_four;

__global__ void kernel(index_t Nx, index_t Ny, value_t r, value_t *u, value_t *u_new)
{
    // grid moves along last index first
    int N_grid_j = (Ny + blockDim.y - 1) / blockDim.y;
    int grid_i = blockIdx.x / N_grid_j;
    int grid_j = blockIdx.x - grid_i * N_grid_j;

    // grid moves along first index first
    // int N_grid_i = (Nx + blockDim.x - 1) / blockDim.x;
    // int grid_j = blockIdx.x / N_grid_i;
    // int grid_i = blockIdx.x - grid_j * N_grid_i;

    int gi = grid_i * blockDim.x + threadIdx.x;
    int gj = grid_j * blockDim.y + threadIdx.y;
    int gid = gi * Ny + gj;

    int ly = threadIdx.y;
    int lx = threadIdx.x;

    int slengthy = blockDim.y + 2;
    // int slengthx = blockDim.x + 2;
    int sx = threadIdx.x + 1;
    int sy = threadIdx.y + 1;
    int sid = sx * slengthy + sy;

    extern __shared__ value_t s_u[]; // (slengthy + 2) * (slengthx + 2)

    if (gi < Nx && gj < Ny)
    {
        s_u[sid] = u[gid];

        if (lx == 0 && gi != 0) // bot
            s_u[sid - slengthy] = u[gid - Ny];

        if (lx == blockDim.x - 1 && gi != Ny - 1) // top
            s_u[sid + slengthy] = u[gid + side_size];

        if (ly == 0 && gj != 0)
        {
            s_u[sid - 1] = u[gid - 1]; // left
            if (lx == 0 && gi != 0)    // left bot corner
                s_u[sid - slengthy - 1] = u[gid - side_size - 1];
            if (lx == blockDim.x - 1 && gi != side_size - 1) // left top corner
                s_u[sid + slengthy - 1] = u[gid + side_size - 1];
        }

        if (ly == blockDim.y - 1 && gj != side_size - 1)
        {
            s_u[sid + 1] = u[gid + 1]; // right
            if (lx == 0 && gi != 0)    // right bot corner
                s_u[sid - slengthy + 1] = u[gid - side_size + 1];
            if (lx == blockDim.x - 1 && gi != side_size - 1) // righ top corner
                s_u[sid + slengthy + 1] = u[gid + side_size + 1];
        }
    }

    __syncthreads();

    if (gi < side_size && gj < side_size)
    {
        if (gi == 0) // i = 0 y = all
        {
            u_new[gid] = c_zero;
        }
        else if (gi == side_size - 1) // i = end y = all
        {
            u_new[gid] = c_zero;
        }
        else
        {
            if (gj == 0) // i = all except for two ends y = 0
            {
                u_new[gid] = c_zero;
            }
            else if (gj == side_size - 1) // i = all except for two ends y = end
            {
                u_new[gid] = c_zero;
            }
            else
            {
                u_new[gid] = (c_one - c_four * r) * s_u[sid] // u_i_j
                             + r * (s_u[sid - slengthy]      // u_i-1_j
                                    + s_u[sid + slengthy]    // u_i+1_j
                                    + s_u[sid - 1]           // u_i_j-1
                                    + s_u[sid + 1]);         // u_i_j+1
            }
        }
    }
}

struct diff2d_cuda_shm : public diff2d<value_t, index_t>
{
    void benchmark()
    {
        print_bench();

        value_t **u = create_array2d<value_t, index_t>(side_size, side_size);
        value_t **u_new = create_array2d<value_t, index_t>(side_size, side_size);

        initial_condition(u, u_new);

        value_t *d_u, *d_u_new;
        value_t *h_u = &u[0][0], *h_u_new = &u_new[0][0];
        checkCudaErrors(hipMalloc(&d_u, total_size * sizeof(value_t)));
        checkCudaErrors(hipMalloc(&d_u_new, total_size * sizeof(value_t)));

        checkCudaErrors(hipMemcpy(d_u, h_u, total_size * sizeof(value_t), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_u_new, h_u_new, total_size * sizeof(value_t), hipMemcpyHostToDevice));
        value_t zero = 0.0;
        value_t one = 1.0;
        value_t four = 4.0;
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_zero), &zero, sizeof(value_t), 0, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_one), &one, sizeof(value_t), 0, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_four), &four, sizeof(value_t), 0, hipMemcpyHostToDevice));

        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        dim3 block = dim3(block0, block1, 1);
        dim3 grid = calculate_grid<index_t>(block, side_size, side_size);
        int sm_memsize = (block.x + 2) * (block.y + 2) * sizeof(value_t);
        std::cout << "  Block: " << block.x << "(x) X " << block.y << "(y)\n"
                  << "  Grid size: " << grid.x << "\n"
                  << "  Shared memory needed: " << sm_memsize << " Byte\n\n";

        loops = 0;
        auto startcpu = std::chrono::high_resolution_clock::now();
        checkCudaErrors(hipEventRecord(start));
        while ((std::chrono::duration_cast<std::chrono::milliseconds>(
                    std::chrono::high_resolution_clock::now() - startcpu)
                    .count()) < 1000.0 * benchtime)
        {
            kernel<<<grid, block, sm_memsize>>>(side_size, r, d_u, d_u_new);
            checkCudaErrorsAfterKernels

            // slengthyap u and u_new
            value_t *tmp = d_u;
            d_u = d_u_new;
            d_u_new = tmp;
            loops++;
        }
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float du = 0;
        checkCudaErrors(hipEventElapsedTime(&du, start, stop));
        duration = 1.0e-3 * du;

        checkCudaErrors(hipMemcpy(h_u, d_u, total_size * sizeof(value_t), hipMemcpyDeviceToHost));

        value_t t = delta_t * value_t(loops) + t0;
        test_result(u, t);
        print_performance();

        destroy_array2d<value_t, index_t>(u);
        destroy_array2d<value_t, index_t>(u_new);
        checkCudaErrors(hipFree(d_u));
        checkCudaErrors(hipFree(d_u_new));
    }

    diff2d_cuda_shm(int narg, char **arg) : diff2d(narg, arg) {}
};

int main(int narg, char **arg)
{
    check_cuda_device();
    diff2d_cuda_shm test(narg, arg);
    test.benchmark();
}