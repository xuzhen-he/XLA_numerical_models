#include "hip/hip_runtime.h"
#include <chrono>
#include "array2d.h"
#include "cuda_helper.h"
#include "vec_bench.h"

#define value_t float
#define index_t int

#define XPXPY  12

__global__ void kernel(index_t n, value_t *x, value_t *y)
{
    index_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n / 4)
    {
        auto tmp_y = reinterpret_cast<float4 *>(y);
        auto tmp_x = reinterpret_cast<float4 *>(x);
        tmp_y[i].x += tmp_x[i].x - tmp_x[i].x + tmp_x[i].x - tmp_x[i].x + tmp_x[i].x - tmp_x[i].x//
             + tmp_x[i].x - tmp_x[i].x + tmp_x[i].x - tmp_x[i].x + tmp_x[i].x - tmp_x[i].x;

        tmp_y[i].y += tmp_x[i].y - tmp_x[i].y + tmp_x[i].y - tmp_x[i].y + tmp_x[i].y - tmp_x[i].y//
             + tmp_x[i].y - tmp_x[i].y + tmp_x[i].y - tmp_x[i].y + tmp_x[i].y - tmp_x[i].y;
        
        tmp_y[i].z += tmp_x[i].z - tmp_x[i].z + tmp_x[i].z - tmp_x[i].z + tmp_x[i].z - tmp_x[i].z
             + tmp_x[i].z - tmp_x[i].z + tmp_x[i].z - tmp_x[i].z + tmp_x[i].z - tmp_x[i].z;

        tmp_y[i].w += tmp_x[i].w - tmp_x[i].w + tmp_x[i].w - tmp_x[i].w + tmp_x[i].w - tmp_x[i].w
             + tmp_x[i].w - tmp_x[i].w + tmp_x[i].w - tmp_x[i].w + tmp_x[i].w - tmp_x[i].w;
    }

    // in only one thread, process final elements (if there are any)
    index_t remainder = n % 4;
    if (i == n / 4 && remainder != 0)
    {
        while (remainder)
        {
            int i = n - remainder--;
            y[i] += x[i] - x[i] + x[i] - x[i] + x[i] - x[i]
             + x[i] - x[i] + x[i] - x[i] + x[i] - x[i];
        }
    }
}

struct vec_copy : public vec_bench<value_t, index_t>
{
    void benchmark()
    {
        print_bench();

#if XPXPY == 6
        std::cout << "\nSimulation info: 1d vec xpxpy 6\n";
#elif XPXPY == 12
        std::cout << "\nSimulation info: 1d vec xpxpy 12\n";
#elif XPXPY == 20
        std::cout << "\nSimulation info: 1d vec xpxpy 20\n";
#else
#endif

        value_t *x = new value_t[total_size];
        value_t *y = new value_t[total_size];

#pragma omp parallel for
        for (index_t j = 0; j < total_size; j++)
        {
            x[j] = 1.0;
            y[j] = 0.0;
        }

        value_t *d_x, *d_y;
        checkCudaErrors(hipMalloc(&d_x, total_size * sizeof(value_t)));
        checkCudaErrors(hipMalloc(&d_y, total_size * sizeof(value_t)));

        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        checkCudaErrors(hipMemcpy(d_x, x, total_size * sizeof(value_t), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_y, y, total_size * sizeof(value_t), hipMemcpyHostToDevice));

        dim3 blockd3 = dim3(block, 1, 1);
        dim3 grid = calc_grid1d(blockd3, total_size/4);
        std::cout << "  Block: " << blockd3.x << "(x) X " << blockd3.y << "(y)\n"
                  << "  Grid size: " << grid.x << "\n\n";

        loops = 0;
        auto startcpu = std::chrono::high_resolution_clock::now();
        checkCudaErrors(hipEventRecord(start));
        while ((std::chrono::duration_cast<std::chrono::milliseconds>(
                    std::chrono::high_resolution_clock::now() - startcpu)
                    .count()) < 1000.0 * benchtime)
        {
            kernel<<<grid, blockd3>>>(total_size, d_x, d_y);
            checkCudaErrorsAfterKernels;
            loops++;
        }
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float du = 0;
        checkCudaErrors(hipEventElapsedTime(&du, start, stop));
        duration = 1.0e-3 * du;

        checkCudaErrors(hipMemcpy(y, d_y, total_size * sizeof(value_t), hipMemcpyDeviceToHost));

        value_t target = (value_t)loops * (value_t)total_size;
        test_result(y, target);
        print_performance();

        delete[] x;
        delete[] y;
        checkCudaErrors(hipFree(d_x));
        checkCudaErrors(hipFree(d_y));
    }

    vec_copy(int narg, char **arg) : vec_bench<value_t, index_t>(narg, arg)
    {
        memory_transfer_per_loop = 3.0 * sizeof(value_t) * double(total_size) /
                                   (1024.0 * 1024.0 * 1024.0);
    }
};

int main(int narg, char **arg)
{
    check_cuda_device();
    vec_copy test(narg, arg);
    test.benchmark();
}