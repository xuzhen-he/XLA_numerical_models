#include "hip/hip_runtime.h"
#include <chrono>
#include "array2d.h"
#include "cuda_helper.h"
#include "vec_bench.h"

#define value_t float
#define index_t int

__global__ void kernel(index_t n, value_t scale, value_t *x, value_t *y)
{
    index_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n / 4)
    {
        auto tmp_y = reinterpret_cast<float4 *>(y);
        auto tmp_x = reinterpret_cast<float4 *>(x);
        tmp_y[i].x += scale * tmp_x[i].x;
        tmp_y[i].y += scale * tmp_x[i].x;
        tmp_y[i].z += scale * tmp_x[i].x;
        tmp_y[i].w += scale * tmp_x[i].x;
    }

    // in only one thread, process final elements (if there are any)
    index_t remainder = n % 4;
    if (i == n / 4 && remainder != 0)
    {
        while (remainder)
        {
            int i = n - remainder--;
            y[i] += scale * x[i];
        }
    }
}

struct vec_copy : public vec_bench<value_t, index_t>
{
    void benchmark()
    {
        print_bench();

        std::cout << "\nSimulation info: 1d vec axpy\n";

        value_t *x = new value_t[total_size];
        value_t *y = new value_t[total_size];
        value_t a = 1.0;

#pragma omp parallel for
        for (index_t j = 0; j < total_size; j++)
        {
            x[j] = 1.0;
            y[j] = 0.0;
        }

        value_t *d_x, *d_y;
        checkCudaErrors(hipMalloc(&d_x, total_size * sizeof(value_t)));
        checkCudaErrors(hipMalloc(&d_y, total_size * sizeof(value_t)));

        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        checkCudaErrors(hipMemcpy(d_x, x, total_size * sizeof(value_t), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_y, y, total_size * sizeof(value_t), hipMemcpyHostToDevice));

        dim3 blockd3 = dim3(block, 1, 1);
        dim3 grid = calc_grid1d(blockd3, total_size/4);
        std::cout << "  Block: " << blockd3.x << "(x) X " << blockd3.y << "(y)\n"
                  << "  Grid size: " << grid.x << "\n\n";

        loops = 0;
        auto startcpu = std::chrono::high_resolution_clock::now();
        checkCudaErrors(hipEventRecord(start));
        while ((std::chrono::duration_cast<std::chrono::milliseconds>(
                    std::chrono::high_resolution_clock::now() - startcpu)
                    .count()) < 1000.0 * benchtime)
        {
            kernel<<<grid, blockd3>>>(total_size, a, d_x, d_y);
            checkCudaErrorsAfterKernels;
            loops++;
        }
        checkCudaErrors(hipEventRecord(stop));
        checkCudaErrors(hipEventSynchronize(stop));
        float du = 0;
        checkCudaErrors(hipEventElapsedTime(&du, start, stop));
        duration = 1.0e-3 * du;

        checkCudaErrors(hipMemcpy(y, d_y, total_size * sizeof(value_t), hipMemcpyDeviceToHost));

        value_t target = (value_t)loops * (value_t)total_size;
        test_result(y, target);
        print_performance();

        delete[] x;
        delete[] y;
        checkCudaErrors(hipFree(d_x));
        checkCudaErrors(hipFree(d_y));
    }

    vec_copy(int narg, char **arg) : vec_bench<value_t, index_t>(narg, arg)
    {
        memory_transfer_per_loop = 3.0 * sizeof(value_t) * double(total_size) /
                                   (1024.0 * 1024.0 * 1024.0);
    }
};

int main(int narg, char **arg)
{
    check_cuda_device();
    vec_copy test(narg, arg);
    test.benchmark();
}